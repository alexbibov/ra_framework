#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_payloads.h"
#include "ox_wrapper/constants.h"

using namespace optix;
using namespace ox_wrapper;

rtDeclareVariable(unsigned int, payload_type, , "Type of ray payload");
rtDeclareVariable(OxRayRadiancePayload, radiance_payload, rtPayload, );
rtDeclareVariable(OxRayRadiancePayloadSimple, radiance_payload_simple, rtPayload, );
rtDeclareVariable(OxRayRadiancePayloadMonochromatic, radiance_payload_monochromatic, rtPayload, );
rtDeclareVariable(OxRayOcclusionPayload, occlusion_payload, rtPayload, );
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");

RT_PROGRAM void __ox_any_hit__()
{
    switch (static_cast<OxRayPayloadType>(payload_type))
    {
    case OxRayPayloadType::radiance:
        memset(radiance_payload.spectral_radiance, 0, constants::max_spectra_pairs_supported * sizeof(float2));

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case OxRayPayloadType::radiance_simple:
        radiance_payload_simple.spectral_radiance = make_float2(0.f, 0.f);

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case OxRayPayloadType::monochromatic:
        radiance_payload_monochromatic.spectral_radiance = 0.f;

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case OxRayPayloadType::occlusion:
        occlusion_payload.is_occluded = true;

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;
    }

    rtTerminateRay();
}