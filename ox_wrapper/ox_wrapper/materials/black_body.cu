#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_payloads.h"
#include "ox_wrapper/constants.h"

using namespace optix;
using namespace ox_wrapper;

rtDeclareVariable(unsigned int, payload_type, , "Type of ray payload");
rtDeclareVariable(OxRayRadiancePayload, radiance_payload, rtPayload, );
rtDeclareVariable(OxRayRadiancePayloadSimple, radiance_payload_simple, rtPayload, );
rtDeclareVariable(OxRayRadiancePayloadMonochromatic, radiance_payload_monochromatic, rtPayload, );
rtDeclareVariable(OxRayOcclusionPayload, occlusion_payload, rtPayload, );

RT_PROGRAM void __ox_any_hit__()
{
    switch (static_cast<OxRayPayloadType>(payload_type))
    {
    case OxRayPayloadType::radiance:
        memset(radiance_payload.spectral_radiance, 0, constants::max_spectra_pairs_supported * sizeof(float2));
        radiance_payload.depth = make_float2(0.f, 0.f);
        break;

    case OxRayPayloadType::radiance_simple:
        radiance_payload_simple.spectral_radiance = make_float2(0.f, 0.f);
        radiance_payload_simple.depth = make_float2(0.f, 0.f);
        break;

    case OxRayPayloadType::monochromatic:
        radiance_payload_monochromatic.spectral_radiance = 0.f;
        radiance_payload_monochromatic.depth = make_float2(0.f, 0.f);
        break;

    case OxRayPayloadType::occlusion:
        occlusion_payload.is_occluded = true;
        occlusion_payload.depth = make_float2(0.f, 0.f);
        break;
    }

    rtTerminateRay();
}