#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/commons.h"
#include "ox_wrapper/ray_radiance_payload.h"


rtDeclareVariable(float, step_size, , "Ray marching step size");
rtDeclareVariable(unsigned int, max_recursion_depth, , "Maximal depth of recursion for scattering traverse");
typedef rtCallableProgramId<float(optix::float3)> absorption_factor_program_id_type;
typedef rtCallableProgramId<float(optix::float3)> scattering_factor_program_id_type;
typedef rtCallableProgramId<float(optix::float3, optix::float2, optix::float2)> phase_function_program_id_type; 
rtDeclareVariable(absorption_factor_program_id_type, absorption_factor, , );
rtDeclareVariable(scattering_factor_program_id_type, scattering_factor, , );
rtDeclareVariable(phase_function_program_id_type, phase_function, , );
rtDeclareVariable(unsigned int, num_importance_directions, , );

rtDeclareVariable(ox_wrapper::OxRayRadiancePayload, ray_payload, rtPayload, "Current ray payload");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");

rtBuffer<optix::float3, 1> importance_directions_buffer;

/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
 The buffer has to be organized as follows:
 1st element contains number of vectors stored in the buffer, hereinafter denoted by N
 The rest of the buffer must contain at least 6*N floats packed into uints where the first 3 floating point elements
 define position of corresponding vector and 3 subsequent elements identify its direction
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;

using namespace optix;
using namespace ox_wrapper;

RT_PROGRAM void __ox_intersect__(void)
{

}