#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/commons.h"
#include "ox_wrapper/ray_payload.h"



rtDeclareVariable(float, step_size, , "Ray marching step size");
typedef rtCallableProgramId<float(optix::float3)> absorption_factor_program_id_type;
typedef rtCallableProgramId<float(optix::float3)> scattering_factor_program_id_type;
typedef rtCallableProgramId<float(optix::float3, optix::float2, optix::float2)> phase_function_program_id_type; 
rtDeclareVariable(absorption_factor_program_id_type, absorption_factor, , );
rtDeclareVariable(scattering_factor_program_id_type, scattering_factor, , );
rtDeclareVariable(phase_function_program_id_type, phase_function, , );
rtDeclareVariable(unsigned int, num_importance_directions, , );

rtDeclareVariable(ox_wrapper::OxRayPayload, ray_payload, rtPayload, "Current ray payload");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");

rtBuffer<float3, 1> importance_directions_buffer;


using namespace optix;
using namespace ox_wrapper;

RT_PROGRAM void __ox_intersect__(void)
{

}