#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/constants.h"
#include "ox_wrapper/ray_payloads.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(float, step_size, , "Ray marching step size");
rtDeclareVariable(unsigned int, max_recursion_depth, , "Maximal depth of recursion for scattering traverse");
rtDeclareVariable(unsigned int, num_spectra_supported, , "Number of wavelengths in use");
rtDeclareVariable(unsigned int, num_importance_directions, , );

typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> absorption_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> scattering_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, optix::float2, optix::float2, unsigned int)> phase_function_program_id_type; 
rtDeclareVariable(absorption_factor_program_id_type, absorption_factor, , );
rtDeclareVariable(scattering_factor_program_id_type, scattering_factor, , );
rtDeclareVariable(phase_function_program_id_type, phase_function, , );

rtDeclareVariable(ox_wrapper::OxRayRadiancePayload, ray_payload, rtPayload, "Current ray payload");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");
rtDeclareVariable(optix::float3, index, rtLaunchIndex, "Index of the current ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");

/*! Determines importance sampling domain for in-scattering component of the volumetric light
 transport equation. The buffer consists of a number of importance direction records with layout
 of each record defined as follows:
 � record begins by a float2 element that defines spherical coordinates of importance direction vector,
 more precisely float2.x contains inclination and float2.y contains azimuth angle
 � the following are N* more float2 elements with each component (i.e. x- and y-) identifying single spectral
 radiance value in the given importance direction OUTSIDE of medium (therefore, current design only supports
 cases where spectral radiance outside of medium does not depend on spatial location).
 
 *Here N=min(num_spectra_supported, constants::max_spectra_pairs_supported)
*/
rtBuffer<optix::float2, 1> importance_directions_buffer;

/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
 The buffer has to be organized as follows:
 1st element contains number of vectors stored in the buffer, hereinafter denoted by N
 The rest of the buffer must contain at least 6*N floats and N unsigned integers ordered in 9-element
 tuples with the first 3 elements being floating point numbers determining the origin of the 
 corresponding ray, succeeding 3 elements being floating point components of the ray's direction, and
 the last 3 unsigned integers determining three-dimensional index of the ray in its original launch grid
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

using namespace optix;
using namespace ox_wrapper;

__device__ float sign(float x)
{
    return
        x > 0 ? 1.f :
        x < 0 ? -1.f :
        0.f;
}

__device__ void pack_ray_info(float3 origin, float3 direction)
{
    unsigned int offset = atomicAdd(&traverse_backup_buffer[0], 1U);
    offset *= 9;

    traverse_backup_buffer[1 + offset] = __float_as_uint(origin.x);
    traverse_backup_buffer[2 + offset] = __float_as_uint(origin.y);
    traverse_backup_buffer[3 + offset] = __float_as_uint(origin.z);

    traverse_backup_buffer[4 + offset] = __float_as_uint(direction.x);
    traverse_backup_buffer[5 + offset] = __float_as_uint(direction.y);
    traverse_backup_buffer[6 + offset] = __float_as_uint(direction.z);

    traverse_backup_buffer[7 + offset] = index.x;
    traverse_backup_buffer[8 + offset] = index.y;
    traverse_backup_buffer[9 + offset] = index.z;
}

__device__ float3 extract_direction_from_angles(float2 inclination_and_azimuthal_angles)
{
    float z = cosf(inclination_and_azimuthal_angles.x);
    float aux = sinf(inclination_and_azimuthal_angles.x);
    float x = aux*cosf(inclination_and_azimuthal_angles.y);
    float y = aux*sinf(inclination_and_azimuthal_angles.y);
    return make_float3(x, y, z);
}

__device__ float2 extract_angles_from_direction(float3 direction)
{
    float inclination_angle = acosf(direction.z);
    float aux{ direction.x / sqrtf(direction.x*direction.x + direction.y*direction.y) };
    float c = direction.y >= 0 ? 1.f : -1.f;
    float azimuthal_angle = acosf(aux*c) + M_PIf*static_cast<float>(direction.y < 0);

    return make_float2(inclination_angle, azimuthal_angle);
}

__device__ void update_ray_payload(float3 p, float3 p_2, float2 direction_of_interest)
{
    unsigned int const ns{ MIN(constants::max_spectra_pairs_supported, num_spectra_supported) };
    for (unsigned int i = 0U; i < ns; ++i)
    {
        float2 S = make_float2(0.f, 0.f);

        // scattering component is only calculated when scattering is enabled
        for (unsigned int j = 0U; j < num_importance_directions; ++j)
        {
            OxRayRadiancePayload scattered_payload;
            //memset(scattered_payload.spectral_radiance, 0, ns*sizeof(float2));
            scattered_payload.tracing_depth = ray_payload.tracing_depth + 1;
            scattered_payload.aux0 = ray_payload.aux0;
            scattered_payload.aux1 = j * (ns + 1) + 1 + i;
            scattered_payload.aux2 = i;

            float2 importance_direction = importance_directions_buffer[j*(ns + 1)];
            Ray scattered_ray = make_Ray(
                p,
                extract_direction_from_angles(importance_direction),
                static_cast<unsigned int>(OxRayType::scattered), 0.f, step_size);

            rtTrace(ox_entry_node, scattered_ray, scattered_payload);

            S += scattered_payload.spectral_radiance[i]
                * phase_function(p, importance_direction, direction_of_interest, i) * sin(importance_direction.x);
        }

        float2 sigma_S_p_2 = num_importance_directions ? scattering_factor(p_2, i) : make_float2(0.f, 0.f);
        float2 phi = expf(-(absorption_factor(p_2, i) + sigma_S_p_2)*intersection_distance);

        ray_payload.spectral_radiance[i] =
            ray_payload.spectral_radiance[i] * phi
            + S * scattering_factor(p, i)*intersection_distance;
    }
}

RT_PROGRAM void __ox_intersect__(void)
{
    int dS = static_cast<int>(sign(-dot(normal, current_ray.direction)));
    ray_payload.aux0 = MAX(0, static_cast<int>(ray_payload.aux0) + dS);
    float3 p{ current_ray.origin + intersection_distance*current_ray.direction };
    
    if (dS > 0)    // the ray has entered object
    {
        ++ray_payload.tracing_depth;
        if (ray_payload.tracing_depth <= max_recursion_depth)
        {
            Ray subsurface_ray = make_Ray(
                p + __uint_as_float(0x800000)*current_ray.direction,
                current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, step_size);

            rtTrace(ox_entry_node, subsurface_ray, ray_payload);
        }
        
        if(ray_payload.tracing_depth > max_recursion_depth)
        {
            ray_payload.tracing_depth = 0U;
            pack_ray_info(current_ray.origin, current_ray.direction);
        }
    }
    else if (dS < 0)    // the ray has left object
    {
        float3 p_2{ current_ray.origin + intersection_distance * .5f * current_ray.direction };
        float2 direction_of_interest = extract_angles_from_direction(current_ray.direction);

        update_ray_payload(p, p_2, direction_of_interest);

        // we still need to try to keep traversing the ray as there might be more media to discover
        ++ray_payload.tracing_depth;
        if (ray_payload.tracing_depth <= max_recursion_depth)
        {
            Ray next_iteration_ray = make_Ray(
                p + __uint_as_float(0x800000)*current_ray.direction, 
                current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, RT_DEFAULT_MAX);

            rtTrace(ox_entry_node, next_iteration_ray, ray_payload);
        }
    }

    // ray has "touched" object: no operation is required
}

RT_PROGRAM void __ox_miss__(void)
{
    if (ray_payload.aux0 > 0 && ray_payload.tracing_depth <= max_recursion_depth)    // "miss" has happened inside of a medium
    {
        float3 p{ current_ray.origin + intersection_distance * current_ray.direction };
        float3 p_2{ current_ray.origin + intersection_distance * .5f * current_ray.direction };
        float2 direction_of_interest = extract_angles_from_direction(current_ray.direction);

        update_ray_payload(p, p_2, direction_of_interest);

        ++ray_payload.tracing_depth;

        Ray next_iteration_ray = make_Ray(
            p, current_ray.direction,
            static_cast<unsigned int>(OxRayType::unknown), 0.f, step_size);

        rtTrace(ox_entry_node, next_iteration_ray, ray_payload);
    }
}

RT_PROGRAM void __ox_intersect_scattered__(void)
{
    // this shader is only called when scattered ray exits the domain of the medium

    unsigned int spectrum = ray_payload.aux2;
    unsigned int idb_offset = ray_payload.aux1;

    float2 incoming_spectral_radiance = importance_directions_buffer[idb_offset];
    ray_payload.spectral_radiance[spectrum] = incoming_spectral_radiance;
}

RT_PROGRAM void __ox_miss_scattered__(void)
{
    // this shader is only invoked from within the medium

    ++ray_payload.tracing_depth;

    if (ray_payload.tracing_depth <= max_recursion_depth)
    {
        float3 p{ current_ray.origin + intersection_distance * current_ray.direction };
        float3 p_2{ current_ray.origin + intersection_distance * .5f * current_ray.direction };

        Ray next_scattered_ray_iteration = make_Ray(
            p, current_ray.direction,
            static_cast<unsigned int>(OxRayType::scattered), 0.f, 
            ray_payload.tracing_depth < max_recursion_depth ? step_size : RT_DEFAULT_MAX);
        rtTrace(ox_entry_node, next_scattered_ray_iteration, ray_payload);

        unsigned int spectrum = ray_payload.aux2;
        float2 phi = expf(-absorption_factor(p_2, spectrum) * intersection_distance);
        ray_payload.spectral_radiance[spectrum] *= phi;
    }
}

RT_CALLABLE float2 __ox_scattering_material_default_absorption_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(1.f, 1.f);
}

RT_CALLABLE float2 __ox_scatterin_material_default_scattering_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(1.f, 1.f);
}

RT_CALLABLE float3 __ox_scattering_material_default_phase_funciton__(float3 pos,
    float2 incident_direction, float3 scattering_direction, unsigned int spectrum)
{
    return make_float2(1.f, 1.f) / (4.f*M_PIf);
}