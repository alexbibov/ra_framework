#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/commons.h"
#include "ox_wrapper/ray_radiance_payload.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(float, step_size, , "Ray marching step size");
rtDeclareVariable(unsigned int, max_recursion_depth, , "Maximal depth of recursion for scattering traverse");
rtDeclareVariable(unsigned int, max_scattering_depth, , "Maximal number of steps allowed for scattered rays");
rtDeclareVariable(unsigned int, num_spectra_supported, , "Number of wavelengths in use");
rtDeclareVariable(unsigned int, num_importance_directions, , );

typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> absorption_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> scattering_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, optix::float2, optix::float2, unsigned int)> phase_function_program_id_type; 
rtDeclareVariable(absorption_factor_program_id_type, absorption_factor, , );
rtDeclareVariable(scattering_factor_program_id_type, scattering_factor, , );
rtDeclareVariable(phase_function_program_id_type, phase_function, , );

rtDeclareVariable(ox_wrapper::OxRayRadiancePayload, ray_payload, rtPayload, "Current ray payload");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");
rtDeclareVariable(optix::float3, index, rtLaunchIndex, "Index of the current ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");


rtBuffer<optix::float2, 1> importance_directions_buffer;

/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
 The buffer has to be organized as follows:
 1st element contains number of vectors stored in the buffer, hereinafter denoted by N
 The rest of the buffer must contain at least 6*N floats and N unsigned integers ordered in 9-element
 tuples with the first 3 elements being floating point numbers determining the origin of the 
 corresponding ray, succeeding 3 elements being floating point components of the ray's direction, and
 the last 3 unsigned integers determining three-dimensional index of the ray in its original launch grid
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

using namespace optix;
using namespace ox_wrapper;

__device__ float sign(float x)
{
    return
        x > 0 ? 1.f :
        x < 0 ? -1.f :
        0.f;
}

__device__ void pack_ray_info(float3 origin, float3 direction)
{
    unsigned int offset = atomicAdd(&traverse_backup_buffer[0], 1U);
    offset *= 9;

    traverse_backup_buffer[1 + offset] = __float_as_uint(origin.x);
    traverse_backup_buffer[2 + offset] = __float_as_uint(origin.y);
    traverse_backup_buffer[3 + offset] = __float_as_uint(origin.z);

    traverse_backup_buffer[4 + offset] = __float_as_uint(direction.x);
    traverse_backup_buffer[5 + offset] = __float_as_uint(direction.y);
    traverse_backup_buffer[6 + offset] = __float_as_uint(direction.z);

    traverse_backup_buffer[7 + offset] = index.x;
    traverse_backup_buffer[8 + offset] = index.y;
    traverse_backup_buffer[9 + offset] = index.z;
}

__device__ float3 extract_direction_from_angles(float2 inclination_and_azimuthal_angles)
{
    float z = cosf(inclination_and_azimuthal_angles.x);
    float aux = sinf(inclination_and_azimuthal_angles.x);
    float x = aux*cosf(inclination_and_azimuthal_angles.y);
    float y = aux*sinf(inclination_and_azimuthal_angles.y);
    return make_float3(x, y, z);
}

__device__ float2 extract_angles_from_direction(float3 direction)
{
    float inclination_angle = acosf(direction.z);
    float aux{ direction.x / sqrtf(direction.x*direction.x + direction.y*direction.y) };
    float c = direction.y >= 0 ? 1.f : -1.f;
    float azimuthal_angle = acosf(aux*c) + M_PIf*static_cast<float>(direction.y < 0);

    return make_float2(inclination_angle, azimuthal_angle);
}

RT_PROGRAM void __ox_intersect__(void)
{
    int const dS = sign(-dot(normal, current_ray.direction));
    ray_payload.aux0 = MAX(0, static_cast<int>(ray_payload.aux0) + dS);
    float3 const p{ current_ray.origin + intersection_distance*current_ray.direction };
    unsigned int const ns{ MIN(OX_MAX_SPECTRA_PAIRS_SUPPORTED, num_spectra_supported) };

    if (dS > 0)    // the ray has entered object
    {
        unsigned int tracing_depth = ray_payload.tracing_depth + 1;

        if (tracing_depth <= max_recursion_depth)
        {
            OxRayRadiancePayload new_ray_payload{ ray_payload };
            new_ray_payload.tracing_depth = tracing_depth;
            new_ray_payload.aux = static_cast<unsigned int>(scattering_state);

            Ray new_ray = make_Ray(
                p + __uint_as_float(0x800000)*current_ray.direction,
                current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, step_size);

            rtTrace(ox_entry_node, new_ray, new_ray_payload);
            memcpy(ray_payload.spectral_radiant_exitance, new_ray_payload.spectral_radiant_exitance, sizeof(float2)*ns);
        }
        else
        {
            pack_ray_info(current_ray.origin, current_ray.direction);
        }
    }
}

RT_PROGRAM void __ox_intersect_scattered__(void)
{

}

RT_PROGRAM void __ox_miss__(void)
{
    if (ray_payload.trace_depth <= max_recursion_depth && ray_payload.aux > 0)
    {
        unsigned int const ns{ MIN(OX_MAX_SPECTRA_PAIRS_SUPPORTED, num_spectra_supported) };
        float3 p{ current_ray.origin + intersection_distance*current_ray.direction };
        float3 p_2{ current_ray.origin + intersection_distance / 2.f*current_ray.direction };
       
        // Compute scattering component when scattering is enabled
        if (num_importance_directions)
        {
            float2 direction_of_interest = extract_angles_from_direction(current_ray.direction);
            float2 S[OX_MAX_SPECTRA_PAIRS_SUPPORTED] = {};
            for (unsigned int j = 0U; j < num_importance_directions; ++j)
            {
                OxRayRadiancePayload scattered_payload{ ray_payload };
                ++scattered_payload.trace_depth;

                float2 importance_direction = importance_directions_buffer[j];
                Ray scattered_ray = make_Ray(
                    p,
                    extract_direction_from_angles(importance_direction),
                    static_cast<unsigned int>(OxRayType::scattered), 0.f, step_size
                );

                rtTrace(ox_entry_node, scattered_ray, scattered_payload);
                for (unsigned int k = 0; k < ns; ++k)
                    S[k] += scattered_payload.spectral_radiand_exitance[k] * phase_function(p, importance_direction, direction_of_interest) * sin(importance_direction.x);
            }
        }

        for (unsigned int i = 0U; i < ns; ++i)
        {
            float2 sigma_A = absorption_factor(p_2, i);
            float2 sigma_S = num_importance_directions ? scattering_factor(p_2, i) : make_float2(0.f, 0.f);

            float2 tau = expf(-(sigma_A + sigma_S)*intersection_distance);
            
            
        }
    }
}