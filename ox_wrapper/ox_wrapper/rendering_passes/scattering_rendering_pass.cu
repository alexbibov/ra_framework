#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/constants.h"
#include "ox_wrapper/ray_payloads.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(float, step_size, , "Ray marching step size");
rtDeclareVariable(uint3, problem_size, , "Original size of the problem");
rtDeclareVariable(optix::uint, max_recursion_depth, , "Maximal depth of recursion for scattering traverse");
rtDeclareVariable(optix::uint, num_spectra_pairs_supported, , "Number of wavelengths in use");
rtDeclareVariable(optix::uint, num_importance_directions, , );

typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> absorption_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> scattering_factor_program_id_type;
typedef rtCallableProgramId<optix::float2(optix::float3, optix::float2, optix::float2, unsigned int)> phase_function_program_id_type; 
rtDeclareVariable(absorption_factor_program_id_type, absorption_factor, , );
rtDeclareVariable(scattering_factor_program_id_type, scattering_factor, , );
rtDeclareVariable(phase_function_program_id_type, phase_function, , );

rtDeclareVariable(ox_wrapper::OxRayRadiancePayload, ray_payload, rtPayload, "Current ray payload");
rtDeclareVariable(ox_wrapper::OxRayRadiancePayloadSimple, ray_payload_scattered, rtPayload, "Payload of the current scattering ray");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");
rtDeclareVariable(optix::uint3, index, rtLaunchIndex, "Index of the current ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");

/*! Determines importance sampling domain for in-scattering component of the volumetric light
 transport equation. The buffer consists of a number of importance direction records with layout
 of each record defined as follows:
 � record begins by a float2 element that defines spherical coordinates of importance direction vector,
 more precisely float2.x stores inclination and float2.y stores azimuthal angle
 � the following are N more float2 elements, where each component (i.e. x- and y-) identifies single spectral
 radiance value in the given importance direction OUTSIDE of medium (therefore, current design only supports
 cases where spectral radiance outside of the medium does not depend on spatial location).
 
 *Here N=min(num_spectra_pairs_supported, constants::max_spectra_pairs_supported)
*/
rtBuffer<optix::float2, 1> importance_directions_buffer;

/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
 The buffer has to be organized as follows:
 1st element contains number of vectors stored in the buffer, hereinafter denoted by N
 The rest of the buffer must contain at least 6*N floats and 3*N unsigned integers ordered in 9-element
 tuples with the first 3 elements being floating point numbers determining the origin of the 
 corresponding ray, succeeding 3 elements being floating point components of the ray's direction, and
 the last 3 unsigned integers determining three-dimensional index of the ray in its original launch grid
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

using namespace optix;
using namespace ox_wrapper;

__device__ uint3 linear_index_to_3d_index(unsigned int idx)
{
    unsigned int layer_idx = idx / (problem_size.x*problem_size.y);
    unsigned int aux = idx - layer_idx * problem_size.x*problem_size.y;
    
    unsigned int scanline_idx = aux / problem_size.x;
    unsigned int ray_idx = aux - scanline_idx * problem_size.x;

    return optix::make_uint3(ray_idx, scanline_idx, layer_idx);
}

__device__ float sign(float x)
{
    return
        x > 0 ? 1.f :
        x < 0 ? -1.f :
        0.f;
}

__device__ void pack_ray_info(float3 origin, float3 direction, uint3 idx)
{
    unsigned int offset = atomicAdd(&traverse_backup_buffer[0], 1U);
    offset *= 9;

    traverse_backup_buffer[1 + offset] = __float_as_uint(origin.x);
    traverse_backup_buffer[2 + offset] = __float_as_uint(origin.y);
    traverse_backup_buffer[3 + offset] = __float_as_uint(origin.z);

    traverse_backup_buffer[4 + offset] = __float_as_uint(direction.x);
    traverse_backup_buffer[5 + offset] = __float_as_uint(direction.y);
    traverse_backup_buffer[6 + offset] = __float_as_uint(direction.z);

    traverse_backup_buffer[7 + offset] = idx.x;
    traverse_backup_buffer[8 + offset] = idx.y;
    traverse_backup_buffer[9 + offset] = idx.z;
}

__device__ float3 extract_direction_from_angles(float2 inclination_and_azimuthal_angles)
{
    float z = cosf(inclination_and_azimuthal_angles.x);
    float aux = sinf(inclination_and_azimuthal_angles.x);
    float x = aux*cosf(inclination_and_azimuthal_angles.y);
    float y = aux*sinf(inclination_and_azimuthal_angles.y);
    return make_float3(x, y, z);
}

__device__ float2 extract_angles_from_direction(float3 direction)
{
    float inclination_angle = acosf(direction.z);
    float aux{ direction.x / sqrtf(direction.x*direction.x + direction.y*direction.y) };
    float c = direction.y >= 0 ? 1.f : -1.f;
    float azimuthal_angle = acosf(aux*c) + M_PIf*static_cast<float>(direction.y < 0);

    return make_float2(inclination_angle, azimuthal_angle);
}

__device__ void update_ray_payload(float3 p, float3 p_2, float2 direction_of_interest, float step)
{
    unsigned int const ns{ MIN(constants::max_spectra_pairs_supported, num_spectra_pairs_supported) };
    for (unsigned int i = 0U; i < ns; ++i)
    {
        float2 S = make_float2(0.f, 0.f);

        // scattering component is only calculated when scattering is enabled
        for (int j = 0; j < num_importance_directions; ++j)
        {
            OxRayRadiancePayloadSimple scattered_payload;
            scattered_payload.spectral_radiance = make_float2(0.f, 0.f);
            scattered_payload.tracing_depth_and_aux =
                make_uint4(
                    ray_payload.tracing_depth_and_aux.x + 1,
                    ray_payload.tracing_depth_and_aux.y,
                    j * (ns + 1) + 1 + i,
                    i);

            float2 importance_direction = importance_directions_buffer[j*(ns + 1)];
            Ray scattered_ray = make_Ray(
                p,
                extract_direction_from_angles(importance_direction),
                static_cast<unsigned int>(OxRayType::scattered), 0.f, step_size);

            rtTrace(ox_entry_node, scattered_ray, scattered_payload);

            S += scattered_payload.spectral_radiance
                * phase_function(p, importance_direction, direction_of_interest, i) * sin(importance_direction.x);
        }

        float2 sigma_S_p_2 = num_importance_directions ? scattering_factor(p_2, i) : make_float2(0.f, 0.f);
        float2 phi = expf(-(absorption_factor(p_2, i) + sigma_S_p_2)*step);

        ray_payload.spectral_radiance[i] =
            ray_payload.spectral_radiance[i] * phi
            + S * scattering_factor(p, i)*step;
    }
}

RT_PROGRAM void __ox_closest_hit__(void)
{
    int dS = static_cast<int>(sign(-dot(normal, current_ray.direction)));
    ray_payload.tracing_depth_and_aux.y = MAX(0, static_cast<int>(ray_payload.tracing_depth_and_aux.y) + dS);
    float3 p{ current_ray.origin + intersection_distance*current_ray.direction };

    
    if (dS > 0)    // the ray has entered object
    {
        ray_payload.depth.x = intersection_distance;
        ray_payload.depth.y = intersection_distance;

        ++ray_payload.tracing_depth_and_aux.x;
        if (ray_payload.tracing_depth_and_aux.x <= max_recursion_depth)
        {
            float const correction = step_size*1e-2f;

            Ray subsurface_ray = make_Ray(
                p + correction*current_ray.direction,
                current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, step_size + correction);

            rtTrace(ox_entry_node, subsurface_ray, ray_payload);
        }
        else
        {
            ray_payload.tracing_depth_and_aux.x = 0U;

            uint3 idx = ray_payload.tracing_depth_and_aux.z > 0 ?
                linear_index_to_3d_index(ray_payload.tracing_depth_and_aux.z - 1) : index;
            // ray_payload.tracing_depth_and_aux.z = 0U;

            pack_ray_info(current_ray.origin, current_ray.direction, idx);
        }
    }
    else if (dS < 0)    // the ray has left object
    {
        ray_payload.depth.y += intersection_distance;

        float3 p_2{ current_ray.origin + intersection_distance * .5f * current_ray.direction };
        float2 direction_of_interest = extract_angles_from_direction(current_ray.direction);

        update_ray_payload(p, p_2, direction_of_interest, intersection_distance);

        // we still need to try to keep traversing the ray as there might be more media to discover
        ++ray_payload.tracing_depth_and_aux.x;
        if (ray_payload.tracing_depth_and_aux.x <= max_recursion_depth)
        {
            float const correction = step_size * 1e-2f;

            Ray next_iteration_ray = make_Ray(
                p + correction*current_ray.direction,
                current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, RT_DEFAULT_MAX);

            rtTrace(ox_entry_node, next_iteration_ray, ray_payload);
        }
    }

    // ray has "touched" object: no operation is required
}

RT_PROGRAM void __ox_miss__(void)
{
    if (ray_payload.tracing_depth_and_aux.y >= 1)    // "miss" has happened inside of medium
    {
        float3 p{ current_ray.origin + step_size * current_ray.direction };
        float3 p_2{ current_ray.origin + step_size * .5f * current_ray.direction };
        float2 direction_of_interest = extract_angles_from_direction(current_ray.direction);

        ray_payload.depth.y += step_size;
        update_ray_payload(p, p_2, direction_of_interest, step_size);

        if (ray_payload.tracing_depth_and_aux.x <= max_recursion_depth)
        {
            ++ray_payload.tracing_depth_and_aux.x;

            Ray next_iteration_ray = make_Ray(
                p, current_ray.direction,
                static_cast<unsigned int>(OxRayType::unknown), 0.f, step_size);

            rtTrace(ox_entry_node, next_iteration_ray, ray_payload);
        }
        else
        {
            ray_payload.tracing_depth_and_aux.x = 0U;

            uint3 idx = ray_payload.tracing_depth_and_aux.z >= 1 ?
                /*optix::make_uint3(ray_payload.tracing_depth_and_aux.z - 1, 0, 0)*/
                linear_index_to_3d_index(ray_payload.tracing_depth_and_aux.z - 1) : index;
            // ray_payload.tracing_depth_and_aux.z = 0U;

            pack_ray_info(p, current_ray.direction, idx);
        }
    }

}

RT_PROGRAM void __ox_closest_hit_scattered__(void)
{
    // this shader is only called when scattered ray exits the domain of the medium
    unsigned int idb_offset = ray_payload_scattered.tracing_depth_and_aux.z;

    float2 incoming_spectral_radiance = importance_directions_buffer[idb_offset];
    ray_payload_scattered.spectral_radiance = incoming_spectral_radiance;
}

RT_PROGRAM void __ox_miss_scattered__(void)
{
    // this shader is only invoked from within the medium

    ++ray_payload_scattered.tracing_depth_and_aux.x;

    if (ray_payload_scattered.tracing_depth_and_aux.x <= max_recursion_depth)
    {
        float3 p{ current_ray.origin + step_size * current_ray.direction };
        float3 p_2{ current_ray.origin + step_size * .5f * current_ray.direction };

        Ray next_scattered_ray_iteration = make_Ray(
            p, current_ray.direction,
            static_cast<unsigned int>(OxRayType::scattered), 0.f, 
            ray_payload_scattered.tracing_depth_and_aux.x < max_recursion_depth ? step_size : RT_DEFAULT_MAX);
        rtTrace(ox_entry_node, next_scattered_ray_iteration, ray_payload_scattered);

        unsigned int spectrum = ray_payload_scattered.tracing_depth_and_aux.w;
        float2 phi = expf(-absorption_factor(p_2, spectrum) * step_size);
        ray_payload_scattered.spectral_radiance *= phi;
    }
}
