#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::float2, center, ,);
rtDeclareVariable(float, radius, , "radius of circle");
rtDeclareVariable(unsigned int, num_materials, , "number of materials attached to geometry primitive");

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "currently traversed ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");


RT_PROGRAM void __ox_intersect__(int primitive_id)
{
    float2 s{ ray.origin.x, ray.origin.y };
    float2 d{ ray.direction.x, ray.direction.y };
    
    float2 aux{ s - center };
    float a{ dot(aux, aux) - radius*radius };
    float b{ dot(s - center, d) };
    float c{ dot(d, d) };

    float D{ sqrt(b*b - a*c) };
    float t{ fminf((-b - D) / a, (-b + D) / a) };

    for (unsigned int i = 0; i < num_materials; ++i)
    {
        if (rtPotentialIntersection(t))
        {
            float2 p{ aux + t * d };
            p /= norm3df(p.x, p.y, 0.f);
            normal.x = p.x; normal.y = p.y; normal.z = 0.f;

            rtReportIntersection(i);
        }
    }
}

RT_PROGRAM void __ox_aabb__(int primitive_id, float aabb[6])
{
    aabb[0] = center.x - radius;
    aabb[1] = center.y - radius;
    aabb[2] = -1e-5f;
    aabb[3] = center.x + radius;
    aabb[4] = center.y + radius;
    aabb[3] = 1e-5f;
}