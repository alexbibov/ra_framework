#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::float2, center, ,);
rtDeclareVariable(float, radius, , "radius of circle");
rtDeclareVariable(unsigned int, num_materials, , "number of materials attached to geometry primitive");

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "currently traversed ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");


RT_PROGRAM void __ox_intersect__(int primitive_id)
{
    float2 s{ ray.origin.x, ray.origin.y };
    float2 d{ ray.direction.x, ray.direction.y };
    
    float2 aux{ s - center };
    float a{ dot(d, d) };
    float b{ dot(aux, d) };
    float c{ dot(aux, aux) - radius * radius };

    float D{ b*b - a*c };
    if (D > 0)
    {
        D = sqrt(D);
        float t1 = (-b - D) / a;
        float t2 = (-b + D) / a;
        
        for (unsigned int i = 0; i < num_materials; ++i)
        {
            bool check_second{ true };

            if (rtPotentialIntersection(t1))
            {
                float2 p{ aux + t1 * d };
                p /= norm3df(p.x, p.y, 0.f);
                normal.x = p.x; normal.y = p.y; normal.z = 0.f;

                if (rtReportIntersection(i))
                    check_second = false;
            }
            
            if (check_second)
            {
                if (rtPotentialIntersection(t2))
                {
                    float2 p{ aux + t2 * d };
                    p /= norm3df(p.x, p.y, 0.f);
                    normal.x = p.x; normal.y = p.y; normal.z = 0.f;

                    rtReportIntersection(i);
                }
            }
        }
    }
}

RT_PROGRAM void __ox_aabb__(int primitive_id, float aabb[6])
{
    aabb[0] = center.x - radius;
    aabb[1] = center.y - radius;
    aabb[2] = -1e-2f;
    aabb[3] = center.x + radius;
    aabb[4] = center.y + radius;
    aabb[5] = 1e-2f;
}