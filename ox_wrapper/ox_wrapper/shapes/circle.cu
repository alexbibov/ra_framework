#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::float2, center, , "position of center of circle");
rtDeclareVariable(float, radius, , "radius of circle");
rtDeclareVariable(unsigned int, num_materials, , "number of materials attached to geometry primitive");

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "currently traversed ray");


RT_PROGRAM void __ox_intersect__(int primitive_id)
{
    float2 s{ ray.origin.x, ray.origin.y };
    float2 d{ ray.direction.x, ray.direction.y };

    float a{ dot(s - center, s - center) - radius*radius };
    float b{ dot(s - center, d) };
    float c{ dot(d, d) };

    float D{ sqrt(b*b - a*c) };
    float t{ fminf((-b - D) / a, (-b + D) / a) };

    if (rtPotentialIntersection(t))
    {
        for (unsigned int i = 0; i < num_materials; ++i)
        {
            rtReportIntersection(i);
        }
    }
}

RT_PROGRAM void __ox_aabb__(int primitive_id, float aabb[6])
{
    aabb[0] = center.x - radius;
    aabb[1] = center.y - radius;
    aabb[2] = -1e-5f;
    aabb[3] = center.x + radius;
    aabb[4] = center.y + radius;
    aabb[3] = 1e-5f;
}