#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/constants.h"
#include "ox_wrapper/ray_payloads.h"

using namespace ox_wrapper;
using namespace optix;


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");
rtDeclareVariable(unsigned int, dimension, , "Dimension of the output buffer");
rtDeclareVariable(unsigned int, payload_type, , "Type of payload");
rtDeclareVariable(unsigned int, ray_type, , "Type of relaunched rays");
rtDeclareVariable(float, ray_parametric_length, , "Parametric length of recasted ray");

rtDeclareVariable(unsigned int, index, rtLaunchIndex, "Thread index");


/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
The buffer has to be organized as follows:
1st element contains number of vectors stored in the buffer, hereinafter denoted by N
The rest of the buffer must contain at least 6*N floats and N unsigned integers ordered in 9-element
tuples with the first 3 elements being floating point numbers determining the origin of the
corresponding ray, succeeding 3 elements being floating point components of the ray's direction, and
the last 3 unsigned integers determining three-dimensional index of the ray in its original launch grid
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;


// supported output buffers
rtBuffer<OxRayRadiancePayload, 1> ox_radiance_payload_1d_out;
rtBuffer<OxRayRadiancePayload, 2> ox_radiance_payload_2d_out;
rtBuffer<OxRayRadiancePayload, 3> ox_radiance_payload_3d_out;

rtBuffer<OxRayRadiancePayloadSimple, 1> ox_radiance_payload_simple_1d_out;
rtBuffer<OxRayRadiancePayloadSimple, 2> ox_radiance_payload_simple_2d_out;
rtBuffer<OxRayRadiancePayloadSimple, 3> ox_radiance_payload_simple_3d_out;

rtBuffer<OxRayRadiancePayloadMonochromatic, 1> ox_radiance_payload_monochromatic_1d_out;
rtBuffer<OxRayRadiancePayloadMonochromatic, 2> ox_radiance_payload_monochromatic_2d_out;
rtBuffer<OxRayRadiancePayloadMonochromatic, 3> ox_radiance_payload_monochromatic_3d_out;

rtBuffer<OxRayOcclusionPayload, 1> ox_occlusion_payload_1d_out;
rtBuffer<OxRayOcclusionPayload, 2> ox_occlusion_payload_2d_out;
rtBuffer<OxRayOcclusionPayload, 3> ox_occlusion_payload_3d_out;


__device__ void unpack_ray(float3& origin, float3& direction, uint3& output_buffer_index)
{
    origin.x = __uint_as_float(traverse_backup_buffer[1 + 9 * index]);
    origin.y = __uint_as_float(traverse_backup_buffer[2 + 9 * index]);
    origin.z = __uint_as_float(traverse_backup_buffer[3 + 9 * index]);

    direction.x = __uint_as_float(traverse_backup_buffer[4 + 9 * index]);
    direction.y = __uint_as_float(traverse_backup_buffer[5 + 9 * index]);
    direction.z = __uint_as_float(traverse_backup_buffer[6 + 9 * index]);

    output_buffer_index.x = traverse_backup_buffer[7 + 9 * index];
    output_buffer_index.y = traverse_backup_buffer[8 + 9 * index];
    output_buffer_index.z = traverse_backup_buffer[9 + 9 * index];
}

RT_PROGRAM void __ox_generate__(void)
{
    float3 origin, direction;
    uint3 output_buffer_index;
    unpack_ray(origin, direction, output_buffer_index);
/*
    ox_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[2].x = origin.x;
    ox_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[2].y = origin.y;
    ox_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[3].x = direction.x;
    ox_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[3].y = direction.y;*/

    Ray current_ray = make_Ray(origin, direction, ray_type, 0.f, ray_parametric_length);

    switch (static_cast<OxRayPayloadType>(payload_type))
    {
    case OxRayPayloadType::radiance:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            OxRayRadiancePayload payload = ox_radiance_payload_1d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            OxRayRadiancePayload payload = ox_radiance_payload_2d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            OxRayRadiancePayload payload = ox_radiance_payload_3d_out[output_buffer_index];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case OxRayPayloadType::radiance_simple:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            OxRayRadiancePayloadSimple payload = ox_radiance_payload_simple_1d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_simple_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            OxRayRadiancePayloadSimple payload = ox_radiance_payload_simple_2d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_simple_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            OxRayRadiancePayloadSimple payload = ox_radiance_payload_simple_3d_out[output_buffer_index];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_simple_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case OxRayPayloadType::monochromatic:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            OxRayRadiancePayloadMonochromatic payload = ox_radiance_payload_monochromatic_1d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_monochromatic_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            OxRayRadiancePayloadMonochromatic payload = ox_radiance_payload_monochromatic_2d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_monochromatic_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            OxRayRadiancePayloadMonochromatic payload = ox_radiance_payload_monochromatic_3d_out[output_buffer_index];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_radiance_payload_monochromatic_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case OxRayPayloadType::occlusion:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            OxRayOcclusionPayload payload = ox_occlusion_payload_1d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_occlusion_payload_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            OxRayOcclusionPayload payload = ox_occlusion_payload_2d_out[idx];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_occlusion_payload_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            OxRayOcclusionPayload payload = ox_occlusion_payload_3d_out[output_buffer_index];
            rtTrace(ox_entry_node, current_ray, payload);
            ox_occlusion_payload_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;
    }
}


