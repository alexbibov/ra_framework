#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_radiance_payload.h"
#include "ox_wrapper/commons.h"


rtDeclareVariable(unsigned int, num_rays, , "Number of casted rays");
rtDeclareVariable(float, emitter_position, , "Position of the emitter");
rtDeclareVariable(float, emitter_size, , "Size of the emitter");
rtDeclareVariable(float, emitter_rotation, , "Rotation of the emitter");
rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(optix::uint1, index, rtLaunchIndex, "Thread index");

rtBuffer<ox_wrapper::OxRayRadiancePayload, 1> ox_output_buffer;

/*! This buffer must be organized as follows: 
 for each generated ray the buffer stores 4*OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED values
 each determining spectral exitance for the corresponding part of the spectrum. All 4*OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED
 elements therefore define the radiant exitance and the buffer in total determines the radiant flux of the emitter.
*/
rtBuffer<float, 1> ox_init_spectral_flux_buffer;

RT_PROGRAM void __ox_generate__(void)
{
    optix::float3 origin{ -emitter_size / 2.f + emitter_size / (num_rays - 1) * index.x, 0.f, 0.f };
    //float3 direction{ 0.f, 1.f, 0.f };

    float c{ cosf(emitter_rotation) }, s{ sinf(emitter_rotation) };
    origin.x = c * origin.x - s * origin.y;
    origin.y = s * origin.x + c * origin.y;

    optix::float3 direction{ -s, c };

    optix::Ray ray = optix::make_Ray(origin, direction, static_cast<unsigned int>(ox_wrapper::OxRayType::unknown), 0.f, RT_DEFAULT_MAX);


    optix::float4 radiant_exitance[OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED];
    for (unsigned int i = 0; i < OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED; ++i)
    {
        radiant_exitance[i].x = ox_init_spectral_flux_buffer[4 * OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED*index.x + 0];
        radiant_exitance[i].y = ox_init_spectral_flux_buffer[4 * OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED*index.x + 1];
        radiant_exitance[i].z = ox_init_spectral_flux_buffer[4 * OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED*index.x + 2];
        radiant_exitance[i].w = ox_init_spectral_flux_buffer[4 * OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED*index.x + 3];
    }

    ox_wrapper::OxRayRadiancePayload payload{};
    memcpy(payload.radiant_exitance, radiant_exitance, sizeof(optix::float4)*OX_MAX_SPECTRA_QUADRUPLETS_SUPPORTED);
    payload.tracing_depth = 0U;
    payload.flags = 0U;

    rtTrace(ox_entry_node, ray, payload);
}