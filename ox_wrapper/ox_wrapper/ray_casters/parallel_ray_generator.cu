#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_payloads.h"
#include "ox_wrapper/constants.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(unsigned int, num_rays, , "Number of casted rays");
rtDeclareVariable(float, emitter_position, , "Position of the emitter");
rtDeclareVariable(float, emitter_size, , "Size of the emitter");
rtDeclareVariable(float, emitter_rotation, , "Rotation of the emitter");
rtDeclareVariable(optix::uint, num_spectra_pairs_supported, , "Number of wavelengths in use");

rtDeclareVariable(unsigned int, index, rtLaunchIndex, "Thread index");

rtBuffer<ox_wrapper::OxRayRadiancePayload, 1> ox_output_buffer;

/*! The buffer is organized as follows:
 for each element of the emitter the buffer must contain M = MIN(constants::max_spectra_pairs_supported, num_spectra_pairs_supported)
 float2-elements, where each component (x and y) of each of these elements 
 defines spectral radiant exitance of the corresponding part of the spectrum. 
 All these values together therefore determine radiant exitance of single emission element of the emitter, 
 and the whole buffer determines the corresponding radiant flux
*/
rtBuffer<optix::float2, 1> ox_init_flux_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))

__device__ optix::float2 rotate_point_on_X_axis(float point_to_rotate, float center_of_rotation, optix::float2 cs)
{
    optix::float2 rv = optix::make_float2(
        cs.x * point_to_rotate/*.x - cs.y * point_to_rotate.y*/ + (1 - cs.x)*center_of_rotation/*.x + cs.y*center_of_rotation.y*/,
        cs.y * point_to_rotate/*.x + cs.x * point_to_rotate.y*/ - cs.y*center_of_rotation/*.x + (1 - cs.x)*center_of_rotation.y*/
    );

    return rv;
}

RT_PROGRAM void __ox_generate__(void)
{
    optix::float3 origin{ -emitter_size / 2.f + emitter_size / (num_rays - 1) * index + emitter_position, 0.f, 0.f };
    //float3 direction{ 0.f, 1.f, 0.f };

    optix::float2 cs = optix::make_float2(cosf(emitter_rotation), sinf(emitter_rotation));
    optix::float2 rotated_point = rotate_point_on_X_axis(origin.x, emitter_position, cs);
    origin.x = rotated_point.x; origin.y = rotated_point.y;
    optix::float3 direction{ -cs.y, cs.x };
    optix::Ray ray = optix::make_Ray(origin, direction, static_cast<unsigned int>(ox_wrapper::OxRayType::unknown), 0.f, RT_DEFAULT_MAX);
    
    unsigned int const ns{ MIN(ox_wrapper::constants::max_spectra_pairs_supported, num_spectra_pairs_supported) };

    ox_wrapper::OxRayRadiancePayload payload{};
    memcpy(payload.spectral_radiance, &ox_init_flux_buffer[ns*index], sizeof(optix::float2)*ns);
    payload.tracing_depth_and_aux = make_uint4(0U, 0U, 0U, 0U);

    rtTrace(ox_entry_node, ray, payload);

    ox_output_buffer[index] = payload;
}