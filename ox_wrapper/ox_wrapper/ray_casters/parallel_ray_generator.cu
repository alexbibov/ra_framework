#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_payloads.h"
#include "ox_wrapper/constants.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(unsigned int, num_rays, , "Number of casted rays");
rtDeclareVariable(float, emitter_position, , "Position of the emitter");
rtDeclareVariable(float, emitter_size, , "Size of the emitter");
rtDeclareVariable(float, emitter_rotation, , "Rotation of the emitter");
rtDeclareVariable(optix::uint, num_spectra_pairs_supported, , "Number of wavelengths in use");

rtDeclareVariable(unsigned int, index, rtLaunchIndex, "Thread index");

rtBuffer<ox_wrapper::OxRayRadiancePayload, 1> ox_output_buffer;

/*! The buffer is organized as follows:
 for each element of the emitter the buffer must contain M = MIN(constants::max_spectra_pairs_supported, num_spectra_pairs_supported)
 float2-elements, where each component (x and y) of each of these elements 
 defines spectral radiant exitance of the corresponding part of the spectrum. 
 All these values together therefore determine radiant exitance of single emission element of the emitter, 
 and the whole buffer determines the corresponding radiant flux
*/
rtBuffer<optix::float2, 1> ox_init_flux_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))

__device__ optix::float2 rotate_point_2d(optix::float2 point_to_rotate, optix::float2 center_of_rotation, optix::float2 cs)
{
    point_to_rotate -= center_of_rotation;

    optix::float2 rv = optix::make_float2(
        cs.x * point_to_rotate.x - cs.y * point_to_rotate.y,
        cs.y * point_to_rotate.x + cs.x * point_to_rotate.y
    );

    rv += center_of_rotation;

    return rv;
}

RT_PROGRAM void __ox_generate__(void)
{
    optix::float3 origin{ -emitter_size / 2.f + emitter_size / (num_rays - 1) * index + emitter_position, 0.f, 0.f };

    optix::float2 cs = optix::make_float2(cosf(emitter_rotation), sinf(emitter_rotation));
    optix::float2 rotated_point = rotate_point_2d(optix::float2{ origin.x, origin.y }, optix::float2{ emitter_position, 0 }, cs);
    optix::float2 rotated_tip = rotate_point_2d(optix::float2{ origin.x, 1 }, optix::float2{ emitter_position, 0 }, cs);
    
    origin.x = rotated_point.x; origin.y = rotated_point.y;
    optix::float3 direction = optix::normalize(optix::make_float3(rotated_tip - rotated_point, 0));
    optix::Ray ray = optix::make_Ray(origin, direction, static_cast<unsigned int>(ox_wrapper::OxRayType::unknown), 0.f, RT_DEFAULT_MAX);
    
    unsigned int const ns{ MIN(ox_wrapper::constants::max_spectra_pairs_supported, num_spectra_pairs_supported) };

    ox_wrapper::OxRayRadiancePayload payload{};
    memcpy(payload.spectral_radiance, &ox_init_flux_buffer[ns*index], sizeof(optix::float2)*ns);
    payload.tracing_depth_and_aux = make_uint4(0U, 0U, 0U, 0U);
    rtTrace(ox_entry_node, ray, payload);

    ox_output_buffer[index] = payload;
}