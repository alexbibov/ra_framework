#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ox_wrapper/ray_radiance_payload.h"
#include "ox_wrapper/commons.h"


rtDeclareVariable(rtObject, ox_entry_node, , "Scene entry node");

rtDeclareVariable(unsigned int, num_rays, , "Number of casted rays");
rtDeclareVariable(float, emitter_position, , "Position of the emitter");
rtDeclareVariable(float, emitter_size, , "Size of the emitter");
rtDeclareVariable(float, emitter_rotation, , "Rotation of the emitter");
rtDeclareVariable(unsigned int, num_spectra_supported, , "Number of wavelengths in use");

rtDeclareVariable(unsigned int, index, rtLaunchIndex, "Thread index");

rtBuffer<ox_wrapper::OxRayRadiancePayload, 1> ox_output_buffer;

/*! The buffer is organized as follows:
 for each element of the emitter the buffer must contain M = MIN(OX_MAX_SPECTRA_PAIRS_SUPPORTED, num_spectra_supported)
 float2-elements, where each component (x and y) of each of these elements 
 defines spectral radiant exitance of the corresponding part of the spectrum. 
 All these values together therefore determine radiant exitance of single emission element of the emitter, 
 and the whole buffer determines the corresponding radiant flux
*/
rtBuffer<optix::float2, 1> ox_init_flux_buffer;

#define MIN(a, b) ((a) < (b) ? (a) : (b))

RT_PROGRAM void __ox_generate__(void)
{
    optix::float3 origin{ -emitter_size / 2.f + emitter_size / (num_rays - 1) * index, 0.f, 0.f };
    //float3 direction{ 0.f, 1.f, 0.f };

    float c{ cosf(emitter_rotation) }, s{ sinf(emitter_rotation) };
    origin.x = c * origin.x - s * origin.y;
    origin.y = s * origin.x + c * origin.y;

    optix::float3 direction{ -s, c };

    optix::Ray ray = optix::make_Ray(origin, direction, static_cast<unsigned int>(ox_wrapper::OxRayType::unknown), 0.f, RT_DEFAULT_MAX);
    
    unsigned int const ns{ MIN(OX_MAX_SPECTRA_PAIRS_SUPPORTED, num_spectra_supported) };

    ox_wrapper::OxRayRadiancePayload payload{};
    memcpy(payload.spectral_radiant_exitance, &ox_init_buffer[ns*index], sizeof(optix::float2)*ns);
    payload.tracing_depth = 0U;
    payload.aux0 = payload.aux1 = payload.aux2 = 0U;

    rtTrace(ox_entry_node, ray, payload);

    ox_output_buffer[index] = payload;
}