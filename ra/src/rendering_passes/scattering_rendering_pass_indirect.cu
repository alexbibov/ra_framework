#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "constants.h"
#include "ray_payloads.h"


rtDeclareVariable(rtObject, ra_entry_node, , "Scene entry node");


rtDeclareVariable(float, step_size, , "Ray marching step size");
rtDeclareVariable(optix::uint, max_recursion_depth, , "Maximal depth of recursion for scattering traverse");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");
rtDeclareVariable(optix::Ray, current_ray, rtCurrentRay, "Currently traversed ray");
rtDeclareVariable(ra::RaRayRadiancePayloadSimple, ray_payload_scattered, rtPayload, "Payload of the current scattering ray");

typedef rtCallableProgramId<optix::float2(optix::float3, unsigned int)> factor_program_id_type;
rtDeclareVariable(factor_program_id_type, absorption_factor, , );


using namespace optix;
using namespace ra;


__device__ void update_ray_payload_scattered(float3 p_2)
{
    unsigned int spectrum = ray_payload_scattered.tracing_depth_and_aux.w;
    float2 phi = expf(-absorption_factor(p_2, spectrum) * step_size);
    ray_payload_scattered.spectral_radiance *= phi;
}

RT_PROGRAM void __ra_any_hit__(void)
{
    printf("Closest hit scattered\n");
    // this shader is only called when scattered ray exits the domain of the medium
    /*unsigned int idb_offset = ray_payload_scattered.tracing_depth_and_aux.z;

    float2 incoming_spectral_radiance = importance_directions_buffer[idb_offset];
    ray_payload_scattered.spectral_radiance = incoming_spectral_radiance;*/
}

RT_PROGRAM void __ra_miss__(void)
{
    // this shader is only invoked from within the medium
    // printf("Miss scattered\n");

    ++ray_payload_scattered.tracing_depth_and_aux.x;

    if (ray_payload_scattered.tracing_depth_and_aux.x < max_recursion_depth)
    {
        float3 p{ current_ray.origin + step_size * current_ray.direction };
        float3 p_2{ current_ray.origin + step_size * .5f * current_ray.direction };

        float d = RT_DEFAULT_MAX;

        Ray next_scattered_ray_iteration = make_Ray(
            p, current_ray.direction,
            static_cast<unsigned int>(RaRayType::scattered), 0.f, d);
        rtTrace(ra_entry_node, next_scattered_ray_iteration, ray_payload_scattered);

        update_ray_payload_scattered(p_2);
    }
}