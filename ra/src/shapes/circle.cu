#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::float2, location, , "location of circle");
rtDeclareVariable(float, radius, , "radius of circle");
rtDeclareVariable(unsigned int, ra_num_materials, , "number of materials attached to geometry primitive");

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "currently traversed ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "Normal of the surface being hit");


RT_PROGRAM void __ra_intersect__(int primitive_id)
{
    float2 s{ ray.origin.x, ray.origin.y };
    float2 d{ ray.direction.x, ray.direction.y };
    // d = optix::normalize(d);

    float2 aux{ s - location };
    // float a{ dot(d, d) };
    float b{ dot(aux, d) };
    float c{ dot(aux, aux) - radius * radius };

    float D{ b*b - /*a**/c };
    if (D > 0)
    {
        D = sqrt(D);
        float t1 = (-b - D)/* / a*/;
        float t2 = (-b + D)/* / a*/;
        
        for (unsigned int i = 0; i < ra_num_materials; ++i)
        {
            bool check_second{ true };

            if (rtPotentialIntersection(t1))
            {
                float2 p{ aux + t1 * d };
                p = optix::normalize(p);
                normal.x = p.x; normal.y = p.y; normal.z = 0.f;

                if (rtReportIntersection(i)) check_second = false;
            }
            
            if (check_second)
            {
                if (rtPotentialIntersection(t2))
                {
                    float2 p{ aux + t2 * d };
                    p = optix::normalize(p);
                    normal.x = p.x; normal.y = p.y; normal.z = 0.f;

                    rtReportIntersection(i);
                }
            }
        }
    }
}

RT_PROGRAM void __ra_aabb__(int primitive_id, float aabb[6])
{
    aabb[0] = location.x - radius;
    aabb[1] = location.y - radius;
    aabb[2] = -1e10f;
    aabb[3] = location.x + radius;
    aabb[4] = location.y + radius;
    aabb[5] = 1e-10f;
}