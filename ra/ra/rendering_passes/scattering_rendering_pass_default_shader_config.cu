#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

RT_CALLABLE_PROGRAM float2 __ra_scattering_default_absorption_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(.1f, .1f);
}

RT_CALLABLE_PROGRAM float2 __ra_scattering_default_scattering_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(1.f, 1.f);
}

RT_CALLABLE_PROGRAM float2 __ra_scattering_default_phase_function__(float3 pos,
    float2 incident_direction, float2 scattering_direction, unsigned int spectrum)
{
    return make_float2(1.f, 1.f) / (4.f*M_PIf);
}