#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ra/ray_payloads.h"
#include "ra/constants.h"

using namespace optix;
using namespace ra;

rtDeclareVariable(unsigned int, payload_type, , "Type of ray payload");
rtDeclareVariable(RaRayRadiancePayload, radiance_payload, rtPayload, );
rtDeclareVariable(RaRayRadiancePayloadSimple, radiance_payload_simple, rtPayload, );
rtDeclareVariable(RaRayRadiancePayloadMonochromatic, radiance_payload_monochromatic, rtPayload, );
rtDeclareVariable(RaRayOcclusionPayload, occlusion_payload, rtPayload, );
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Parametric distance from ray origin to the intersection");

RT_PROGRAM void __ra_any_hit__()
{
    switch (static_cast<RaRayPayloadType>(payload_type))
    {
    case RaRayPayloadType::radiance:
        memset(radiance_payload.spectral_radiance, 0, constants::max_spectra_pairs_supported * sizeof(float2));

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case RaRayPayloadType::radiance_simple:
        radiance_payload_simple.spectral_radiance = make_float2(0.f, 0.f);

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case RaRayPayloadType::monochromatic:
        radiance_payload_monochromatic.spectral_radiance = 0.f;

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;

    case RaRayPayloadType::occlusion:
        occlusion_payload.is_occluded = true;

        if (!radiance_payload.depth.x) radiance_payload.depth.x = intersection_distance;
        radiance_payload.depth.y += intersection_distance;
        break;
    }

    rtTerminateRay();
}