#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

#include "ra/constants.h"
#include "ra/ray_payloads.h"

using namespace ra;
using namespace optix;


rtDeclareVariable(rtObject, ra_entry_node, , "Scene entry node");
rtDeclareVariable(unsigned int, dimension, , "Dimension of the output buffer");
rtDeclareVariable(unsigned int, payload_type, , "Type of payload");
rtDeclareVariable(unsigned int, ray_type, , "Type of relaunched rays");
rtDeclareVariable(float, ray_parametric_length, , "Parametric length of recasted ray");
rtDeclareVariable(uint3, problem_size, , "Original size of the problem");

rtDeclareVariable(unsigned int, index, rtLaunchIndex, "Thread index");


/*! Traverse backup is a standard buffer, which enables implementation of multi-pass ray-tracing algorithms.
The buffer has to be organized as follows:
1st element contains number of vectors stored in the buffer, hereinafter denoted by N
The rest of the buffer must contain at least 6*N floats and N unsigned integers ordered in 9-element
tuples with the first 3 elements being floating point numbers determining the origin of the
corresponding ray, succeeding 3 elements being floating point components of the ray's direction, and
the last 3 unsigned integers determining three-dimensional index of the ray in its original launch grid
*/
rtBuffer<unsigned int, 1> traverse_backup_buffer;


// supported output buffers
rtBuffer<RaRayRadiancePayload, 1> ra_radiance_payload_1d_out;
rtBuffer<RaRayRadiancePayload, 2> ra_radiance_payload_2d_out;
rtBuffer<RaRayRadiancePayload, 3> ra_radiance_payload_3d_out;

rtBuffer<RaRayRadiancePayloadSimple, 1> ra_radiance_payload_simple_1d_out;
rtBuffer<RaRayRadiancePayloadSimple, 2> ra_radiance_payload_simple_2d_out;
rtBuffer<RaRayRadiancePayloadSimple, 3> ra_radiance_payload_simple_3d_out;

rtBuffer<RaRayRadiancePayloadMonochromatic, 1> ra_radiance_payload_monochromatic_1d_out;
rtBuffer<RaRayRadiancePayloadMonochromatic, 2> ra_radiance_payload_monochromatic_2d_out;
rtBuffer<RaRayRadiancePayloadMonochromatic, 3> ra_radiance_payload_monochromatic_3d_out;

rtBuffer<RaRayOcclusionPayload, 1> ra_occlusion_payload_1d_out;
rtBuffer<RaRayOcclusionPayload, 2> ra_occlusion_payload_2d_out;
rtBuffer<RaRayOcclusionPayload, 3> ra_occlusion_payload_3d_out;


__device__ void unpack_ray(float3& origin, float3& direction, uint3& output_buffer_index)
{
    origin.x = __uint_as_float(traverse_backup_buffer[1 + 9 * index]);
    origin.y = __uint_as_float(traverse_backup_buffer[2 + 9 * index]);
    origin.z = __uint_as_float(traverse_backup_buffer[3 + 9 * index]);

    direction.x = __uint_as_float(traverse_backup_buffer[4 + 9 * index]);
    direction.y = __uint_as_float(traverse_backup_buffer[5 + 9 * index]);
    direction.z = __uint_as_float(traverse_backup_buffer[6 + 9 * index]);

    output_buffer_index.x = traverse_backup_buffer[7 + 9 * index];
    output_buffer_index.y = traverse_backup_buffer[8 + 9 * index];
    output_buffer_index.z = traverse_backup_buffer[9 + 9 * index];
}

RT_PROGRAM void __ra_generate__(void)
{
    float3 origin, direction;
    uint3 output_buffer_index;
    unpack_ray(origin, direction, output_buffer_index);
/*
    ra_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[2].x = origin.x;
    ra_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[2].y = origin.y;
    ra_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[3].x = direction.x;
    ra_radiance_payload_1d_out[output_buffer_index.x].spectral_radiance[3].y = direction.y;*/

    Ray current_ray = make_Ray(origin, direction, ray_type, 0.f, ray_parametric_length);

    switch (static_cast<RaRayPayloadType>(payload_type))
    {
    case RaRayPayloadType::radiance:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            RaRayRadiancePayload payload = ra_radiance_payload_1d_out[idx];
            payload.tracing_depth_and_aux.z = idx + 1;
            ++payload.tracing_depth_and_aux.w;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            RaRayRadiancePayload payload = ra_radiance_payload_2d_out[idx];
            payload.tracing_depth_and_aux.z = idx.y*problem_size.x + idx.x + 1;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            RaRayRadiancePayload payload = ra_radiance_payload_3d_out[output_buffer_index];
            
            payload.tracing_depth_and_aux.z =
                output_buffer_index.z*problem_size.x*problem_size.y
                + output_buffer_index.y*problem_size.x + output_buffer_index.x + 1;
            
            rtTrace(ra_entry_node, current_ray, payload);

            ra_radiance_payload_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case RaRayPayloadType::radiance_simple:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            RaRayRadiancePayloadSimple payload = ra_radiance_payload_simple_1d_out[idx];
            payload.tracing_depth_and_aux.z = idx + 1;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_simple_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            RaRayRadiancePayloadSimple payload = ra_radiance_payload_simple_2d_out[idx];
            payload.tracing_depth_and_aux.z = idx.y*problem_size.x + idx.x + 1;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_simple_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            RaRayRadiancePayloadSimple payload = ra_radiance_payload_simple_3d_out[output_buffer_index];

            payload.tracing_depth_and_aux.z =
                output_buffer_index.z*problem_size.x*problem_size.y
                + output_buffer_index.y*problem_size.x + output_buffer_index.x + 1;
            
            rtTrace(ra_entry_node, current_ray, payload);
            
            ra_radiance_payload_simple_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case RaRayPayloadType::monochromatic:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            RaRayRadiancePayloadMonochromatic payload = ra_radiance_payload_monochromatic_1d_out[idx];
            payload.tracing_depth_and_aux.z = idx + 1;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_monochromatic_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            RaRayRadiancePayloadMonochromatic payload = ra_radiance_payload_monochromatic_2d_out[idx];
            payload.tracing_depth_and_aux.z = idx.y*problem_size.x + idx.x + 1;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_radiance_payload_monochromatic_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            RaRayRadiancePayloadMonochromatic payload = ra_radiance_payload_monochromatic_3d_out[output_buffer_index];

            payload.tracing_depth_and_aux.z =
                output_buffer_index.z*problem_size.x*problem_size.y
                + output_buffer_index.y*problem_size.x + output_buffer_index.x + 1;

            rtTrace(ra_entry_node, current_ray, payload);
            
            ra_radiance_payload_monochromatic_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;

    case RaRayPayloadType::occlusion:
        switch (dimension)
        {
        case 1:
        {
            unsigned int idx = output_buffer_index.x;
            RaRayOcclusionPayload payload = ra_occlusion_payload_1d_out[idx];
            // payload.tracing_depth_and_aux.z = idx;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_occlusion_payload_1d_out[idx] = payload;
            break;
        }

        case 2:
        {
            uint2 idx = make_uint2(output_buffer_index.x, output_buffer_index.y);
            RaRayOcclusionPayload payload = ra_occlusion_payload_2d_out[idx];
            // payload.tracing_depth_and_aux.z = idx.y*problem_size.x + idx.x;
            rtTrace(ra_entry_node, current_ray, payload);
            ra_occlusion_payload_2d_out[idx] = payload;
            break;
        }

        case 3:
        {
            RaRayOcclusionPayload payload = ra_occlusion_payload_3d_out[output_buffer_index];

            /*payload.tracing_depth_and_aux.z =
                output_buffer_index.z*problem_size.x*problem_size.y
                + output_buffer_index.y*problem_size.x + output_buffer_index.x;*/

            rtTrace(ra_entry_node, current_ray, payload);
            
            ra_occlusion_payload_3d_out[output_buffer_index] = payload;
            break;
        }
        }
        break;
    }
}


