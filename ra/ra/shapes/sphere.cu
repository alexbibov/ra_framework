#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(optix::float3, location, , "location of sphere");
rtDeclareVariable(float, radius, , "radius of sphere");
rtDeclareVariable(unsigned int, ra_num_materials, "number of materials attached to geometry");

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "currently traversed ray");

rtDeclareVariable(optix::float3, normal, attribute attrNormal, "normal vector of surface");
rtDeclareVariable(optix::float3, tangent, attribute attrTangent, "tangent vector of surface");
rtDeclareVariable(optix::float2, texcoord, attribute attrTexcoord, "texture coordinate of surface");


__device__ float nz_sgn(float x)
{
    return x >= 0 ? 1.f : -1.f;
}


__device__ float angle_from_sin_and_cos(float s, float c)
{
    return acosf(c)*nz_sgn(s) + M_PIf * static_cast<float>(s < 0);
}

__device__ void setAttributes(float t)
{
    float3 p{ s + t * d };

    {
        // set normal

        float3 n = optix::normalize(n);
        normal.x = n.x; normal.y = n.y; normal.z = n.z;
    }

    {
        // set tangent

        float l = optix::length(p);

        float cos_theta = p.y / l;
        float sin_theta = sqrtf(1.f - cos_theta * cos_theta);

        float cos_phi = p.x / l / sin_theta;
        float sin_phi = p.z / l / sin_theta;

        tangent.x = -sin_phi;
        tangent.y = 0.f;
        tangent.z = cos_phi;


        // set texture coordinates (for now we use equirectangular projection as it is the simplest)
        float theta = 1 - acosf(cos_theta) / M_PIf;    // it is just more natural this way, so the textures can be drawn from lower to upper latitudes
        float phi = angle_from_sin_and_cos(sin_phi, cos_phi) / (2 * M_PIf);

        texcoord.x = theta;
        texcoord.y = phi;
    }

}


RT_PROGRAM void __ra_intersect__(int primitive_id)
{
    float3 o{ ray.origin.x, ray.origin.y, ray.origin.z };
    float3 d{ ray.direction.x, ray.direction.y, ray.direction.z };
    // d = optix::normalize(d);

    float3 s = o - location;
    float oml2 = optix::dot(s, s);
    float oml_d = optix::dot(s, d);

    float D = oml_d * oml_d - oml2;
    if (D >= 0)
    {
        D = sqrt(D);
        float t1 = oml_d - D;
        float t2 = oml_d + D;

        for (unsigned i = 0U; i < ra_num_materials; ++i)
        {
            bool check_second{ true };

            if (rtPotentialIntersection(t1))
            {
                setAttributes(t1);
                if (rtReportInteresection(i)) check_second = false;
            }

            if (check_second)
            {
                if (rtPotentialIntersection(t2))
                {
                    setAttributes(t2);
                    rtReportIntersection(i);
                }
            }
        }
    }

}

RT_PROGRAM void __ra_aabb__(int primitive_id, float aabb[6])
{
    aabb[0] = location.x - radius;
    aabb[1] = location.y - radius;
    aabb[2] = location.z - radius;
    aabb[3] = location.x + radius;
    aabb[4] = location.y + radius;
    aabb[5] = location.z + radius;
}