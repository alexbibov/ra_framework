#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float2, planet_location, , "Location of the center of the planet");
rtDeclareVariable(float, planet_radius, , "Radius of the planet");
rtDeclareVariable(float, atmosphere_thickness, , "Thickness of the atmosphere");


RT_CALLABLE_PROGRAM float2 exponential_decay_absorption(float3 pos, unsigned int spectrum)
{
    float2 _2d_position = make_float2(pos.x, pos.y) - planet_location;
    float height = length(_2d_position) - planet_radius;

    return height < atmosphere_thickness ? make_float2(atmosphere_thickness-height, 2*(atmosphere_thickness-height)) 
        : make_float2(0.f, 0.f);
}

// RT_CALLABLE_PROGRAM float2 __ox_scattering_default_scattering_factor__(float3 pos, unsigned int spectrum)
// {
//     return make_float2(1.f, 1.f);
// }

// RT_CALLABLE_PROGRAM float2 __ox_scattering_default_phase_function__(float3 pos,
//     float2 incident_direction, float2 scattering_direction, unsigned int spectrum)
// {
//     return make_float2(1.f, 1.f) / (4.f*M_PIf);
// }