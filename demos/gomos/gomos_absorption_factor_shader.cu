#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float, wavelength_buffer_offset, , "Offset in wavelength buffer");
rtDeclareVariable(float, num_wavelengths, , "Total number of wavelengths");
rtDeclareVariable(float, num_heights, , "Number of atmospheric layers");
rtDeclareVariable(float, num_gas_profiles, , "Number of gas profiles");

rtDeclareVariable(float2, planet_location, , "Location of the center of the planet");
rtDeclareVariable(float, planet_radius, , "Radius of the planet");
rtDeclareVariable(float, atmosphere_thickness, , "Thickness of the atmosphere");

rtBuffer<float, 2> gas_profiles;
rtBuffer<float, 2> cross_sections;

RT_CALLABLE_PROGRAM float2 gomos_absorption_lut(float3 pos, unsigned int spectrum)
{
    unsigned height = static_cast<unsigned>(round((pos.y + atmosphere_thickness*.5f)*num_heights) - 1);

    float2 absorption_coefficient = make_float2(0.f, 0.f);
    for(int i = 0; i < static_cast<unsigned>(num_gas_profiles); ++i)
    {
        float X = gas_profiles[make_uint2(height, i)];
        float C1 = 0.f, C2 = 0.f;
        if(2*spectrum + wavelength_buffer_offset < num_wavelengths)
        {
            C1 = cross_sections[make_uint2(static_cast<unsigned>(2*spectrum + wavelength_buffer_offset), i)];
        }
        if(2*spectrum + 1 + wavelength_buffer_offset < num_wavelengths)
        {
            C2 = cross_sections[make_uint2(static_cast<unsigned>(2*spectrum + 1 + wavelength_buffer_offset), i)];
        }
        
       absorption_coefficient += X*make_float2(C1, C2);
    }
    return 400*absorption_coefficient;
}

