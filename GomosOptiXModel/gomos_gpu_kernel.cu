#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "../CUDA/v9.0/include/hip/hip_vector_types.h"
#include "ray_payload.h"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, "Current ray");
rtDeclareVariable(gomos_optix_model::RayPayload, payload, rtPayload, "Ray payload structure");
rtDeclareVariable(float, intersection_distance, rtIntersectionDistance, "Ray intersection distance");

rtDeclareVariable(unsigned int, ray_index, rtLaunchIndex, "Ray generation index");

rtDeclareVariable(float, planet_radius, , "Radius of the planet");
rtDeclareVariable(float, height, , "Height of the atmospheric layer");
rtDeclareVariable(float, density, , "Density of the atmospheric layer");
rtDeclareVariable(unsigned int, recursion_depth, , "Depth of ray tracing recursion");

rtDeclareVariable(rtObject, top_object, , "Scene graph entry object");

rtBuffer<gomos_optix_model::RayPayload> ray_marching_buffer;
rtBuffer<gomos_optix_model::ConvergedRayPayload> converged_rays_buffer;

// Intersection program for the atmospheric layer
RT_PROGRAM void atmospheric_layer_intersection_program(int)
{
    float const R = planet_radius + height;
    float sTd = dot(ray.origin, ray.direction);
    float dTd = dot(ray.direction, ray.direction);
    float sTs = dot(ray.origin, ray.origin);
    float D = sTd*sTd - dTd*(sTs - R*R);

    if (D >= 0)
    {
        float t[2] = { (-sTd - D) / dTd, (-sTd + D) / dTd };
        
        bool check_second_intersection_point{ true };
        if (rtPotentialIntersection(t[0]))
        {
            check_second_intersection_point = !rtReportIntersection(0);
        }
            
        if (check_second_intersection_point && rtPotentialIntersection(t[1]))
        {
            rtReportIntersection(0);
        }
    }
}


// Bounding box program for atmospheric layer
RT_PROGRAM void atmospheric_layer_bbox_program(int, float result[6])
{
    // by default we assume coordinate system with the origin located in the center point of the planet

    float const R = planet_radius + height;
    result[0] = result[1] = result[2] = -R;
    result[3] = result[4] = result[5] = R;
}


// Any hit program for the atmospheric layer
RT_PROGRAM void atmospheric_layer_any_hit_program(void)
{
    rtIgnoreIntersection();    // we are not interested in some random hit between the ray and geometry, only the closest hit matters
}


// Closest hit program for the atmosperic layer
RT_PROGRAM void atmospheric_layer_closest_hit_program(void)
{
    /*payload.intensity = height > 0 ? payload.intensity * optix::expf(-density*intersection_distance) : 0.f;
    float3 next_ray_marching_step = ray.origin + intersection_distance*ray.direction;

    if (payload.generation > 0)
    {
        optix::Ray next_ray{ next_ray_marching_step,  ray.direction, 0, 0.f };
        gomos_optix_model::RayPayload next_payload{ payload.intensity, next_ray_marching_step, --payload.generation };
    }*/
}


// Miss program is used to test ray convergence
RT_PROGRAM void atmosperic_layer_miss_program(void)
{
    //payload.converged = true;
}


// Ray generation program, runs once per iteration of the ray marching
RT_PROGRAM void ray_marching_step(void)
{
    /*float3 new_ray_origin{ output_buffer.x, output_buffer.y, output_buffer.z };
    optix::Ray new_ray{ new_ray_origin, float3{ 0.f, 1.f, 0.f }, 0, 0.f };

    */
}