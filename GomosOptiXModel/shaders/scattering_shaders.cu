#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_vector_functions.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(float2, atmosphere_center, , "Nominal center of atmosphere");
rtDeclareVariable(float, atmosphere_radius, , "Nominal radius of atmosphere");

RT_CALLABLE_PROGRAM float2 __ox_scattering_material_default_absorption_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(1.f, 1.f);
}

RT_CALLABLE_PROGRAM float2 __ox_scattering_material_default_scattering_factor__(float3 pos, unsigned int spectrum)
{
    return make_float2(1.f, 1.f);
}

RT_CALLABLE_PROGRAM float2 __ox_scattering_material_default_phase_funciton__(float3 pos,
    float2 incident_direction, float3 scattering_direction, unsigned int spectrum)
{
    return make_float2(1.f, 1.f) / (4.f*M_PIf);
}